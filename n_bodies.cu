#include "hip/hip_runtime.h"
#include "config.h"
#include "types.h"

#include <cstdio>
#define BODY_OFFSET (BODY_SINGLE_VARS + dims * BODY_REPEATED_VARS)
#define BODY(ARR, X) (ARR + (X * BODY_OFFSET))
#define MASS(BODY) (*(BODY + 0))
#define RADI(BODY) (*(BODY + 1))
#define POS(BODY) (BODY + BODY_SINGLE_VARS)
#define ACC(BODY) (BODY + dims + BODY_SINGLE_VARS)
#define VEL(BODY) (BODY + 2 * dims + BODY_SINGLE_VARS)
// macro usage - MASS(BODY(bodies, i)) or POS(BODY(bodies, i))[1] (this gives y)
extern "C" {
__device__
void body_interaction(K* self, K* other, int dims) {
    K dist = EPS2;
    for (int i = 0; i < dims; ++i) {
        K axis = POS(other)[i] - POS(self)[i];
        dist += axis * axis;
    }
    dist = dist * dist * dist;
    dist = K(1) / sqrtf(dist);
    dist = MASS(other) * dist;
    for (int i = 0; i < dims; ++i) {
        ACC(self)[i] += (POS(other)[i] - POS(self)[i]) * dist;
    }
}

__global__
void brute_calculate_interactions(K* bodies, int n, int dims) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x >= n) return;
    for (int i = 0; i < n; i++) {
        body_interaction(BODY(bodies, x), BODY(bodies, i), dims);
    }
}

__global__
void update_velocity(K* bodies, int n, K tick, int dims) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x >= n) return;
    for (int i = 0; i < dims; ++i) {
        VEL(BODY(bodies, x))[i] += ACC(BODY(bodies, x))[i] * tick;
    }
}

__global__
void advance_bodies(K* bodies, int n, K tick, int dims) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x >= n) return;
    for (int i = 0; i < dims; ++i) {
        POS(BODY(bodies, x))[i] += VEL(BODY(bodies, x))[i] * tick;
    }
}

__global__
void set_zero_to_acceleration(K* bodies, int n, int dims) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x >= n) return;
    for (int i = 0; i < dims; ++i) {
        ACC(BODY(bodies, x))[i] = 0;
    }
}

__device__
bool is_nearby(K* self, K* other, int dims) {
    K dist = 0;
    for (int i = 0; i < dims; ++i) {
        K axis = POS(other)[i] - POS(self)[i];
        dist += axis * axis;
    }
    K sq_r = RADI(other) + RADI(self);
    sq_r *= sq_r;
    return dist < sq_r;
}

__global__
void glue_nearby(K* bodies, int n, int dims) {
    // TODO(laiqu) implement this as proper Kernel.
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            if (i == j) continue;
            if (is_nearby(BODY(bodies, i), BODY(bodies, j), dims) &&
                    MASS(BODY(bodies, i)) >= MASS(BODY(bodies, j))) {
               MASS(BODY(bodies, i)) += MASS(BODY(bodies, j));
               MASS(BODY(bodies, j)) = 0;
               RADI(BODY(bodies, i)) += RADI(BODY(bodies, j));
               RADI(BODY(bodies, j)) = 0;
            }
        }
    }
}
}
