#include "hip/hip_runtime.h"
#include "config.h"

#include <cstdio>
#define BODY_OFFSET (1 + dims * 2)
#define BODY(ARR, X) (ARR + (X * BODY_OFFSET))
#define MASS(BODY) (*(BODY + 0))
#define POS(BODY) (BODY + 1)
#define ACC(BODY) (BODY + dims + 1)
// macro usage - MASS(BODY(bodies, i)) or POS(BODY(bodies, i))[1] (this gives y)
extern "C" {
__device__
void body_interaction(K* self, K* other, int dims) {
    K dist = EPS2;
    for (int i = 0; i < dims; ++i) {
        K axis = POS(other)[i] - POS(self)[i];
        dist += axis * axis;
    }
    dist = dist * dist * dist;
    dist = K(1) / sqrtf(dist);
    dist = MASS(other) * dist;
    for (int i = 0; i < dims; ++i) {
        ACC(self)[i] += (POS(other)[i] - POS(self)[i]) * dist;
    }
}

__global__
void brute_calculate_interactions(K* bodies, int n, int dims) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= n || y >= n) return;
    body_interaction(BODY(bodies, x), BODY(bodies, y), dims);
}

__global__
void advance_bodies(K* bodies, int n, K tick, int dims) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= n || y >= n) return;
    for (int i = 0; i < dims; ++i) {
        POS(BODY(bodies, x))[i] += ACC(BODY(bodies, y))[i] * tick;
    }
}
}
