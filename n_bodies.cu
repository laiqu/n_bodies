#include "hip/hip_runtime.h"
#include "config.h"

#include <cstdio>
#define BODY_OFFSET (1 + dims * 3)
#define BODY(ARR, X) (ARR + (X * BODY_OFFSET))
#define MASS(BODY) (*(BODY + 0))
#define POS(BODY) (BODY + 1)
#define ACC(BODY) (BODY + dims + 1)
#define VEL(BODY) (BODY + 2 * dims + 1)
// macro usage - MASS(BODY(bodies, i)) or POS(BODY(bodies, i))[1] (this gives y)
extern "C" {
__device__
void body_interaction(K* self, K* other, int dims) {
    K dist = EPS2;
    for (int i = 0; i < dims; ++i) {
        K axis = POS(other)[i] - POS(self)[i];
        dist += axis * axis;
    }
    dist = dist * dist * dist;
    dist = K(1) / sqrtf(dist);
    dist = MASS(other) * dist;
    for (int i = 0; i < dims; ++i) {
        ACC(self)[i] += (POS(other)[i] - POS(self)[i]) * dist;
    }
}

__global__
void brute_calculate_interactions(K* bodies, int n, int dims) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x >= n) return;
    for (int i = 0; i < n; i++) {
        body_interaction(BODY(bodies, x), BODY(bodies, i), dims);
    }
}

__global__
void update_velocity(K* bodies, int n, K tick, int dims) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x >= n) return;
    for (int i = 0; i < dims; ++i) {
        VEL(BODY(bodies, x))[i] += ACC(BODY(bodies, x))[i] * tick;
    }
}

__global__
void advance_bodies(K* bodies, int n, K tick, int dims) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x >= n) return;
    for (int i = 0; i < dims; ++i) {
        POS(BODY(bodies, x))[i] += VEL(BODY(bodies, x))[i] * tick;
    }
}

__global__
void set_zero_to_acceleration(K* bodies, int n, int dims) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x >= n) return;
    for (int i = 0; i < dims; ++i) {
        ACC(BODY(bodies, x))[i] = 0;
    }
}
}
